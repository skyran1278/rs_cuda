
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > n)
        return;

    y[index] = x[index] + y[index];
}

int main(void)
{
    int N = 1000000;
    float *x, *y;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);
    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }

    std::cout << "Max error: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}
